#include "hip/hip_runtime.h"
/*###############################################################################
#
# Project:  Open Source RS Programming Starter Book
# Purpose:  Satellite image processing functions examples
# Author:   Yann Chemin, <yann.chemin@gmail.com>
#
###############################################################################
# Copyright (c) 2008, Yann Chemin <yann.chemin@gmail.com>
#
# Permission is hereby granted, free of charge, to any person obtaining a
# copy of this software and associated documentation files (the "Software"),
# to deal in the Software without restriction, including without limitation
# the rights to use, copy, modify, merge, publish, distribute, sublicense,
# and/or sell copies of the Software, and to permit persons to whom the
# Software is furnished to do so, subject to the following conditions:
#
# The above copyright notice and this permission notice shall be included
# in all copies or substantial portions of the Software.
#
# THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
# OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
# FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
# THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
# LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
# FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
# DEALINGS IN THE SOFTWARE.
###############################################################################*/
#include <stdio.h>
#include <gdal.h>
#include <stdlib.h>

__global__ void waterid(float *red, float *nir, float *b7, float *water, int nXS)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	if ( i < nXS ){
		water[i]=100.f*(1.f+((nir[i]-red[i])/(red[i]+nir[i])));
		if(water[i] < 0.1 && b7[i] < 0.04)
			water[i] = 1;
		else
			water[i] = 0;	
	}
}

int main()
{
	/**GDAL STUFF***************/
	int err; /*catch errors*/
	//Loading the input files
	GDALAllRegister();
	GDALDatasetH hD1 = GDALOpen("b1.tif",GA_ReadOnly);
	GDALDatasetH hD2 = GDALOpen("b2.tif",GA_ReadOnly);
	GDALDatasetH hD3 = GDALOpen("b7.tif",GA_ReadOnly);

	if(hD1==NULL||hD2==NULL||hD3==NULL){
		printf("At least one input file could not be loaded\n");
		exit(EXIT_FAILURE);
	}

	//Loading the file infos 
	GDALDriverH hDr1 = GDALGetDatasetDriver(hD1);
	char **options = NULL;
	options = CSLSetNameValue( options, "TILED", "YES" );
	options = CSLSetNameValue( options, "COMPRESS", "DEFLATE" );
	options = CSLSetNameValue( options, "PREDICTOR", "2" );
	//Creating output file
	GDALDatasetH hDOut = GDALCreateCopy( hDr1, "water.tif",hD1,FALSE,options,NULL,NULL);
	GDALRasterBandH hBOut = GDALGetRasterBand(hDOut,1);
	
	//Loading the file bands 
	GDALRasterBandH hB1 = GDALGetRasterBand(hD1,1);
	GDALRasterBandH hB2 = GDALGetRasterBand(hD2,1);
	GDALRasterBandH hB3 = GDALGetRasterBand(hD3,1);
	
	int nX = GDALGetRasterBandXSize(hB1);
	int nY = GDALGetRasterBandYSize(hB1);

	/**CUDA STUFF***************/
	int N=nX; 	
	/* Allocate Unified Memory arrays (CUDA 6+)*/
	float *red, *nir, *b7, *water;
	hipMallocManaged(&red, N*sizeof(float));
	hipMallocManaged(&nir, N*sizeof(float));
	hipMallocManaged(&b7, N*sizeof(float));
	hipMallocManaged(&water, N*sizeof(float));
	
	for(int i=0; i<N;i++)
		water[i]=0.0;

	/* Compute the Blocks of data to be sent to GPU */
	// On GeForce 8600 Galaxy x=256
	// On GeForce 9500 Galaxy & GeForce 9800 GT x=512
	int x=512;
	dim3 dimBlock(x);
	dim3 dimGrid ( (N / dimBlock.x) + (!(N % dimBlock.x)?0:1)) ;
	
	//Accessing the data rowxrow
	for(int row=0;row<nY;row++){
		/* Read input files through GDAL */		
		err = GDALRasterIO(hB1,GF_Read,0,row,nX,1,red,nX,1,GDT_Float32,0,0);
		err = GDALRasterIO(hB2,GF_Read,0,row,nX,1,nir,nX,1,GDT_Float32,0,0);
		err = GDALRasterIO(hB3,GF_Read,0,row/2,nX/2,1,b7,nX,1,GDT_Float32,0,0);
		/* Add arrays red, nir, b7 and store result in water */
		waterid<<<dimGrid,dimBlock>>>(red, nir, b7, water, N);
		/* Block until device completed processing */
		hipDeviceSynchronize();
		/* Write output file through GDAL */
		err = GDALRasterIO(hBOut,GF_Write,0,row,nX,1,water,nX,1,GDT_Float32,0,0);
	}
	/* Free the memory */
	hipFree(red); 
	hipFree(nir);
	hipFree(b7);
	hipFree(water);
	GDALClose(hD1);
	GDALClose(hD2);
	GDALClose(hD3);
	GDALClose(hDOut);
}
 

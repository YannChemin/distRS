#include "hip/hip_runtime.h"
/*###############################################################################
#
# Project:  Open Source RS Programming Starter Book
# Purpose:  Satellite image processing functions examples
# Author:   Yann Chemin, <yann.chemin@gmail.com>
#
###############################################################################
# Copyright (c) 2008, Yann Chemin <yann.chemin@gmail.com>
#
# Permission is hereby granted, free of charge, to any person obtaining a
# copy of this software and associated documentation files (the "Software"),
# to deal in the Software without restriction, including without limitation
# the rights to use, copy, modify, merge, publish, distribute, sublicense,
# and/or sell copies of the Software, and to permit persons to whom the
# Software is furnished to do so, subject to the following conditions:
#
# The above copyright notice and this permission notice shall be included
# in all copies or substantial portions of the Software.
#
# THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
# OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
# FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
# THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
# LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
# FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
# DEALINGS IN THE SOFTWARE.
###############################################################################*/
#include <stdio.h>
#include <gdal.h>
#include <stdlib.h>

__global__ void ndviid(float *red, float *nir, float *ndvi, int nXS)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	if ( i < nXS ){
		ndvi[i]=100.f*(1.f+((nir[i]-red[i])/(red[i]+nir[i])));
	}
}

int main()
{
	/**GDAL STUFF***************/
	int err; /*catch errors*/
	//Loading the input files
	GDALAllRegister();
	GDALDatasetH hD1 = GDALOpen("b1.tif",GA_ReadOnly);
	GDALDatasetH hD2 = GDALOpen("b2.tif",GA_ReadOnly);

	if(hD1==NULL||hD2==NULL){
		printf("At least one input file could not be loaded\n");
		exit(EXIT_FAILURE);
	}

	//Loading the file infos 
	GDALDriverH hDr1 = GDALGetDatasetDriver(hD1);

	//Creating output file 
	GDALDatasetH hDOut = GDALCreateCopy( hDr1, "ndvi.tif",hD1,FALSE,NULL,NULL,NULL);
	GDALRasterBandH hBOut = GDALGetRasterBand(hDOut,1);
	
	//Loading the file bands 
	GDALRasterBandH hB1 = GDALGetRasterBand(hD1,1);
	GDALRasterBandH hB2 = GDALGetRasterBand(hD2,1);
	
	int nX = GDALGetRasterBandXSize(hB1);
	int nY = GDALGetRasterBandYSize(hB1);

	/**CUDA STUFF***************/
	int N=nX; 	
	/* Allocate Unified Memory arrays (CUDA 6+)*/
	float *red, *nir, *ndvi;
	hipMallocManaged(&red, N*sizeof(float));
	hipMallocManaged(&nir, N*sizeof(float));
	hipMallocManaged(&ndvi, N*sizeof(float));
	
	for(int i=0; i<N;i++)
		ndvi[i]=0.0;

	/* Compute the Blocks of data to be sent to GPU */
	// On GeForce 8600 Galaxy x=256
	// On GeForce 9500 Galaxy & GeForce 9800 GT x=512
	int x=512;
	dim3 dimBlock(x);
	dim3 dimGrid ( (N / dimBlock.x) + (!(N % dimBlock.x)?0:1)) ;
	
	//Accessing the data rowxrow
	for(int row=0;row<nY;row++){
		/* Read input files through GDAL */		
		err = GDALRasterIO(hB1,GF_Read,0,row,nX,1,red,nX,1,GDT_Float32,0,0);
		err = GDALRasterIO(hB2,GF_Read,0,row,nX,1,nir,nX,1,GDT_Float32,0,0);
		/* Add arrays red, nir, b7 and store result in water */
		ndviid<<<dimGrid,dimBlock>>>(red, nir, ndvi, N);
		/* Block until device completed processing */
		hipDeviceSynchronize();
		/* Write output file through GDAL */
		err = GDALRasterIO(hBOut,GF_Write,0,row,nX,1,ndvi,nX,1,GDT_Float32,0,0);
	}
	/* Free the memory */
	hipFree(red); 
	hipFree(nir);
	hipFree(ndvi);
	GDALClose(hD1);
	GDALClose(hD2);
	GDALClose(hDOut);
}
 
